#include "hip/hip_runtime.h"
#ifndef _Ptensors0_cu
#define _Ptensors0_cu

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include <thrust/tuple.h>

#include "Ptens_base.hpp"
#include "RtensorPack.hpp"
#include "AindexPack.hpp"
#include "CSRmatrix.hpp"
//#include "Ptensors0.hpp"
//#include "Rtensor2_view.hpp"
//#include "Rtensor3_view.hpp"
//#include "Itensor1_view.hpp"
//#include "Itensor2_view.hpp"
//#include "CUDAhelpers.hpp"


__global__ void Ptensors0_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  rarr[rdir[2*i]+c]+=xarr[xdir[2*i]+c];
}


__global__ void Ptensors0_reduce0_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const int* xiarr, const int* xidir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  const int tix=xiarr[xidir[2*i]];
  rarr[rdir[2*i]+c]+=xarr[xdir[2*tix]+c];
}


__global__ void Ptensors0_broadcast0_kernel(float* xarr, const int* xdir, const float* rarr, const int* rdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;
  xarr[xdir[2*i]+c]+=rarr[rdir[2*i]+c];
}


__global__ void Ptensors0_broadcast0_kernel(float* xarr, const int* xdir, const int* xiarr, const int* xidir, const float* rarr, const int* rdir, const int* bmap){
  const int b=blockIdx.x;
  const int c=threadIdx.x;

  const int boffs=bmap[3*b];
  const int N=bmap[3*b+1];
  const int target=bmap[3*b+2];

  float t=0;
  for(int j=0; j<N; j++){
    const int src=bmap[boffs+2*j];
    const float w=*reinterpret_cast<const float*>(bmap+boffs+2*j+1);
    t+=w*rarr[rdir[2*src]+c];
  }
  xarr[xdir[2*target]+c]+=t;
}


__global__ void Ptensors0_gather_kernel(float* rarr, const int* rdir, const float* xarr, const int* xdir, const float* marr, const int* mdir){
  const int i=blockIdx.x;
  const int c=threadIdx.x;

  const int moffs=mdir[2*i];
  const int N=mdir[2*i+1]/2;
  float t=0;
  for(int j=0; j<N; j++){
    const int jix=*reinterpret_cast<const int*>(marr+moffs+2*j);
    const int jweight=marr[moffs+2*j+1];
    t+=jweight*xarr[xdir[2*jix]+c];
  }
  rarr[rdir[2*i]+c]+=t;
}


// -----------------------------------------------------------------------------------------------------------


namespace ptens{

  void Ptensors0_reduce0_cu(cnine::RtensorPack& R, const cnine::RtensorPack& x, int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    int N=R.size();
    Ptensors0_reduce0_kernel<<<N,n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev));
  }

  void Ptensors0_reduce0_cu(cnine::RtensorPack& R, const cnine::RtensorPack& x, const AindexPack& list, int offs, int n, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(list.dev==1);
    int N=R.size();
    Ptensors0_reduce0_kernel<<<N,n,0,stream>>>(R.arrg,R.dir.garr(dev),x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev));
  }

  void Ptensors0_broadcast0_cu(cnine::RtensorPack& x, const cnine::RtensorPack& R, const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    Ptensors0_broadcast0_kernel<<<R.size(),x.dim_of(0,0),0,stream>>>
      (x.arrg+offs,x.dir.garr(dev),R.arrg,R.dir.garr(dev));
  }

  void Ptensors0_broadcast0_cu(cnine::RtensorPack& x, const cnine::RtensorPack& R, const AindexPack& list, const int offs, const hipStream_t& stream){
    int dev=R.dev;
    PTENS_ASSRT(R.dev==1);
    PTENS_ASSRT(x.dev==1);
    PTENS_ASSRT(list.dev==1);
    Ptensors0_broadcast0_kernel<<<R.size(),x.dim_of(0,0),0,stream>>>
      (x.arrg+offs,x.dir.garr(dev),list.arrg,list.dir.garr(dev),R.arrg,R.dir.garr(dev),list.get_barr(1));
  }

  void Ptensors0_gather_cu(cnine::RtensorPack& r, const cnine::RtensorPack& x, const cnine::CSRmatrix<float>& gmap, const hipStream_t& stream){
    int dev=r.dev;
    PTENS_ASSRT(r.dev==1);
    PTENS_ASSRT(x.dev==1);
    const_cast<cnine::CSRmatrix<float>&>(gmap).to_device(dev);
    Ptensors0_gather_kernel<<<r.size(),x.dim_of(0,0),0,stream>>>
      (r.arrg,r.dir.garr(dev),x.arrg,x.dir.garr(dev),gmap.arrg,gmap.dir.garr(dev));
  }

}

#endif 
